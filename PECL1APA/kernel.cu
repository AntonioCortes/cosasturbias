
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <windows.h>
#include "book.h"

#define AZUL 1
#define ROJO 2
#define NARANJA 3
#define VERDE 4
#define MARRON 5
#define AMARILLO 6

void juego(int filas, int columnas, int num_colores, bool cargar_partida, FILE *& datos_partida, FILE *& archivo_matriz);
void generar_matriz(int *& matriz, long tam_matriz, int num_colores);
void dibujar_matriz(int * matriz, int filas, int columnas);
void cargar_datos(int &dificultad, int &filas, int &columnas, FILE *& datos_partida); //carga los datos de la partida (dificultad, numero filas y columnas, pero no la matriz)
void cargar_matriz(int *& matriz, long tam_matriz, FILE *& archivo_matriz);
void guardar_partida(int * matriz, int dificultad, int filas, int columnas,FILE *& archivo_matriz, FILE *& datos_partida);//guarda los datos de la partida (dificultad, numero de filas y columnas) en un archivo .txt y la matriz en un archivo .data
void comprobar_dimensiones(int filas, int columnas, bool & dimensiones_adecuadas);//comprueba si las dimensiones del tableropermiten correr en un bloque SM

int main(int argc, char ** argv)
{
	int dificultad = 0;
	int filas = 0;
	int columnas = 0;
	int opcion = 0;
	FILE * datos_partida;
	FILE  * archivo_matriz;
	bool dimensiones_adecuadas = false;

	//SetConsoleDisplayMode(GetStdHandle(STD_OUTPUT_HANDLE), CONSOLE_FULLSCREEN_MODE, 0);
	srand(time(NULL));

	printf("Seleccione 1 para empezar una nueva partida o 2 para cargar partida\n");
	scanf("%i", &opcion);

	if (opcion == 1)
	{
		do
		{
			printf("Introduzca nivel de dificultad (1 o 2): ");
			scanf("%i", &dificultad);

			if ((dificultad != 1) && (dificultad != 2))
			{
				printf("Error al elegir nivel de dificultad\n");
				system("cls");
			}

		} while ((dificultad != 1) && (dificultad != 2));

		do
		{
			printf("Introduzca el numero de filas y columnas del tablero\n");
			printf("Filas: ");
			scanf("%i", &filas);
			printf("Columnas: ");
			scanf("%i", &columnas);

			comprobar_dimensiones(filas, columnas, dimensiones_adecuadas);

			if (!dimensiones_adecuadas)
			{
				printf("Error, el tablero es demasiado grande para correr en un bloque SM\n");
				system("pause");
				system("cls");
			}
		} while (!dimensiones_adecuadas);

		juego(filas, columnas, dificultad, false, datos_partida, archivo_matriz);
	}
	else if (opcion == 2)
	{		
		cargar_datos(dificultad, filas, columnas, datos_partida);
		juego(filas, columnas, dificultad, true, datos_partida, archivo_matriz);
	}

	system("pause");
	return 0;
}

void juego(int filas,int columnas, int dificultad, bool cargar_partida, FILE *& datos_partida, FILE *& archivo_matriz)
{

	bool salir = false;
	int opcion = 0;
	int num_colores = (dificultad == 1) ? 5 : 6;
	long tam_matriz = filas * columnas;
	int  * matriz = (int *) malloc(tam_matriz * sizeof(int));
	int pos_fila = 0;
	int pos_columna = 0;
	
	if (cargar_partida)
	{
		cargar_matriz(matriz, tam_matriz, archivo_matriz);
	}
	else
	{
		generar_matriz(matriz, tam_matriz, num_colores);
	}

	while (!salir)
	{
		system("cls");
		printf("dificuldad = %i\tfilas = %i\tcolumnas = %i\n", dificultad, filas, columnas);
		printf(" ----------------------\n"
			   "| 1 = seguir jugando   |\n"
			   "| 2 = guardar partida  |\n"
			   "| 3 = salir del juego  |\n"
			   " ---------------------- \n\n");
		dibujar_matriz(matriz, filas, columnas);

		printf("Elija opcion: ");
		scanf("%i", &opcion);

		switch (opcion)
		{
			case 1:
			{
				printf("Introduzca posicion de la casilla (fila/columna):\n");
				printf("fila: ");
				scanf("%i", &pos_fila);
				printf("columna: ");
				scanf("%i", &pos_columna);

				/*
				do
				{
					//jugar()
				} while (hay_ceros());
				*/

				
				break;
			}
			case 2:
			{				
				guardar_partida(matriz, dificultad, filas, columnas, archivo_matriz, datos_partida);
				break;
			}
			case 3:
			{
				salir = true;
				break;
			}
			default:
				break;
		}
	}
}

void comprobar_dimensiones(int filas, int columnas, bool & dimensiones_adecuadas)
{
	hipDeviceProp_t propiedades_gpu;
	hipGetDeviceProperties(&propiedades_gpu, 0);

	long capacidad_sm = propiedades_gpu.maxThreadsDim[0] * propiedades_gpu.maxThreadsDim[1] * propiedades_gpu.maxThreadsDim[2];
	long tam_matriz =  filas * columnas;

	dimensiones_adecuadas = (tam_matriz > capacidad_sm) ? false : true;
}

void generar_matriz(int *& matriz, long tam_matriz, int num_colores)
{
	for (int i = 0; i < tam_matriz; i++)
	{
		matriz[i] = rand() % num_colores + 1;
	}
}

void dibujar_matriz(int * matriz, int filas, int columnas)
{
	int valor = 0;

	printf(" \t");

	for (int i = 0; i < columnas; i++)
	{
		printf("%i   ", i);
	}
	printf("\n\n\n");

	for (int i = 0; i < filas; i++)
	{
		printf("%i\t", i);
		for (int n = 0; n < columnas; n++)
		{
			valor = matriz[i * columnas + n];

			switch (valor) 
			{
			case 0:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 0);
				break;
			case 1:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 1);
				break;
			case 2:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 4);
				break;
			case 3:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13);
				break;
			case 4:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 2);
				break;
			case 5:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 6);
				break;
			case 6:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14);
				break;
			}

			printf("%i   ", valor);
		}
		printf("\n");
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
	}

}

void guardar_partida(int * matriz, int dificultad, int filas, int columnas, FILE *& archivo_matriz, FILE *& datos_partida)
{
	long tam_matriz = filas * columnas; 

	if ((datos_partida = fopen("datos_partida.txt", "w")) == NULL)
	{
		printf("error abriendo el archivo \"datos_partida.txt\" \n");
		system("pause");
		exit(1);
	}
	else
	{
		fprintf(datos_partida, "%i\n", dificultad);
		fprintf(datos_partida, "%i\n", filas);
		fprintf(datos_partida, "%i", columnas);
		fclose(datos_partida);
	}

	if ((archivo_matriz = fopen("matriz.data", "wb")) == NULL)
	{
		printf("error abriendo el archivo \"archivo_matriz.txt\" \n");
		system("pause");
		exit(1);
	}
	else
	{
		fwrite(matriz, sizeof(int), tam_matriz, archivo_matriz);
		fclose(archivo_matriz);
	}
}



void cargar_datos(int &dificultad, int &filas, int &columnas, FILE *& datos_partida)
{
	if ((datos_partida = fopen("datos_partida.txt", "r")))
	{
		int linea_actual = 0;
		while (!feof(datos_partida))
		{
			linea_actual++;

			switch (linea_actual)
			{
			case 1:
			{
					  fscanf(datos_partida, "%d", &dificultad);
					  break;
			}
			case 2:
			{
					  fscanf(datos_partida, "%d", &filas);
					  break;
			}
			case 3:
			{
					  fscanf(datos_partida, "%d", &columnas);
					  break;
			}
			default:
				break;
			}
		}

		fclose(datos_partida);
	}
	else
	{
		printf("error abriendo el archivo datos_partida.txt\n");
		return;
	}
}

void cargar_matriz(int *& matriz, long tam_matriz, FILE *& archivo_matriz)
{
	if ((archivo_matriz = fopen("matriz.data", "rb")))
	{
		fread(matriz, sizeof(int), tam_matriz, archivo_matriz);
	}
	else
	{
		printf("error abriendo el archivo \"archivo_matriz\"");
		return;
	}
	fclose(archivo_matriz);
}

