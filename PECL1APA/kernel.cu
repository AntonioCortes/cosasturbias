
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>

#define AZUL 1
#define ROJO 2
#define NARANJA 3
#define VERDE 4
#define MARRON 5
#define AMARILLO 6

void juego(const int filas, const int columnas, int num_colores, bool cargar_partida);
void generar_matriz(int *& matriz, int tam_matriz, int num_colores);

int main(int argc, char ** argv)
{
	int dificultad = 0;
	int filas = 0;
	int columnas = 0;
	int opcion = 0;

	srand(time(NULL));

	printf("Seleccione 1 para empezar una nueva partida o 2 para cargar partida\n");
	scanf("%i", &opcion);

	if (opcion == 1)
	{
		do
		{
			printf("Introduzca nivel de dificultad (1 o 2): ");
			scanf("%i", &dificultad);

			if ((dificultad != 1) && (dificultad != 2))
			{
				printf("Error al elegir nivel de dificultad\n");
				system("cls");
			}

		} while ((dificultad != 1) && (dificultad != 2));

		printf("Introduzca el numero de filas y columnas del tablero\n");
		printf("Filas: ");
		scanf("%i", &filas);
		printf("Columnas: ");
		scanf("%i", &columnas);

		juego(filas, columnas, dificultad, false);
	}
	else if (opcion == 2)
	{
		//TODO: recoger los datos del txt e inicializar los valores de filas columnas y dificultad
		filas = 0;
		columnas = 0;
		dificultad = 0;

		printf("cargar partida\n");
		juego(filas, columnas, dificultad, true);
	}


	system("pause");
	return 0;
}

void juego(const int filas,const int columnas, int dificultad, bool cargar_partida)
{

	bool salir = false;
	int opcion = 0;
	int num_colores = (dificultad == 1) ? 5 : 6;
	const int tam_matriz = filas * columnas;
	int  * matriz = (int *) malloc(tam_matriz * sizeof(int));

	if (cargar_partida)
	{
		//TODO:recoger la matriz serializada deserializarla y guardarla en la variable matriz
		//matriz = 
	}
	else
	{
		generar_matriz(matriz, tam_matriz, num_colores);

		//comprobar los valores de la matriz
		for (int i = 0; i < tam_matriz; i++)
		{
			printf("%i\n",matriz[i]);
		}
		scanf("%i", opcion);
	}

	while (!salir)
	{
		system("cls");
		printf(" ----------------------\n"
			   "| 1 = seguir jugando   |\n"
			   "| 2 = guardar partida  |\n"
			   "| 3 = salir del juego  |\n"
			   " ---------------------- \n");

		scanf("%i", &opcion);

		switch (opcion)
		{
			case 1:
				break;
			case 2:
				break;
			case 3:
			{
				salir = true;
				break;
			}
			default:
				break;
		}
	}
}

void generar_matriz(int *& matriz, int tam_matriz, int num_colores)
{
	for (int i = 0; i < tam_matriz; i++)
	{
		matriz[i] = rand() % num_colores + 1;
	}
}