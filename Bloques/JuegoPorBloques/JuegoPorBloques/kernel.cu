#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <windows.h>
//#include "book.h"

#define AZUL 1
#define ROJO 2
#define NARANJA 3
#define VERDE 4
#define MARRON 5
#define AMARILLO 6
#define BOMBAHOR 7
#define BOMBAVER 8
#define BOMBATNT 9
#define BOMBAPUZZLE 10
#define TILE_WIDTH 8

void juego(int filas, int columnas, int num_colores, bool cargar_partida, FILE *& datos_partida, FILE *& archivo_matriz);
void generar_matriz(int *& matriz, long tam_matriz, int num_colores);
void dibujar_matriz(int * matriz, int filas, int columnas);
void cargar_datos(int &dificultad, int &filas, int &columnas, FILE *& datos_partida); //carga los datos de la partida (dificultad, numero filas y columnas, pero no la matriz)
void cargar_matriz(int *& matriz, long tam_matriz, FILE *& archivo_matriz);
void guardar_partida(int * matriz, int dificultad, int filas, int columnas, FILE *& archivo_matriz, FILE *& datos_partida);//guarda los datos de la partida (dificultad, numero de filas y columnas) en un archivo .txt y la matriz en un archivo .data
void comprobar_dimensiones(int filas, int columnas, bool & dimensiones_adecuadas);//comprueba si las dimensiones del tableropermiten correr en un bloque SM
void jugar(int *tablero, int fil, int col, int size, int fila, int columna, int num_colores);
void generarAleatorios(int *& matriz, int tam_matriz, int num_colores);
bool es_bomba(int * matriz, int fila, int columna, int num_columnas, int &tipo_bomba);//comprueba si la posici�n elegida por el jugador corresponde a una bomba
void explotar_vertical(int *& tablero, long tam_tablero, int filas, int columnas, int columna);//helper que ejecuta el kernel de la explosion de la bomba vertical
void explotar_horizontal(int *& tablero, long tam_tablero, int filas, int columnas, int fila);//helper que ejecuta el kernel de la explosion de la bomba horizontal
void explotar_tnt(int *& tablero, long tam_tablero, int filas, int columnas, int fila, int columna);

__global__ void KernelJugar(int *tablero, int fila, int columna, int i, int j, int bomba, int color);
__global__ void explosion_vertical(int * tablero, int anchura_tablero, int columna);
__global__ void explosion_horizontal(int * tablero, int anchura_tablero, int fila);
__global__ void explosion_tnt(int * tablero, long tam_tablero, int filas, int columnas, int fila, int columna);
__device__ void comprobarBloques(int *tablero, int x, int y, int fila, int columna);
__device__ void comprobarBloquesArriba(int *tablero, int x, int y, int fila, int columna);
__device__ void comprobarBloquesDerecha(int *tablero, int x, int y, int fila, int columna);
__device__ void comprobarBloquesIzquierda(int *tablero, int x, int y, int fila, int columna);
__device__ void comprobarBloquesAbajo(int *tablero, int x, int y, int fila, int columna);
__device__ void borrarArriba(int *tablero, int x, int y, int fila, int columna);
__device__ void borrarAbajo(int *tablero, int x, int y, int fila, int columna);
__device__ void borrarDerecha(int *tablero, int x, int y, int fila, int columna);
__device__ void borrarIzquierda(int *tablero, int x, int y, int fila, int columna);
//__device__ void bombaVertical(int *tablero, int x, int y, int fila, int columna);
//__device__ void bombaHorizontal(int *tablero, int x, int y, int fila, int columna);
//__device__ void bombaTNT(int *tablero, int x, int y, int fila, int columna);
__device__ void bombaPuzzle(int *tablero, int x, int y, int fila, int columna, int color);


int main(int argc, char ** argv)
{
	int dificultad = 0;
	int filas = 0;
	int columnas = 0;
	int opcion = 0;
	FILE * datos_partida;
	FILE  * archivo_matriz;
	bool dimensiones_adecuadas = false;

	//SetConsoleDisplayMode(GetStdHandle(STD_OUTPUT_HANDLE), CONSOLE_FULLSCREEN_MODE, 0);
	srand(time(NULL));

	printf("Seleccione 1 para empezar una nueva partida o 2 para cargar partida\n");
	scanf("%i", &opcion);

	if (opcion == 1)
	{
		do
		{
			printf("Introduzca nivel de dificultad (1 o 2): ");
			scanf("%i", &dificultad);

			if ((dificultad != 1) && (dificultad != 2))
			{
				printf("Error al elegir nivel de dificultad\n");
				system("cls");
			}

		} while ((dificultad != 1) && (dificultad != 2));

		do
		{
			printf("Introduzca el numero de filas y columnas del tablero\n");
			printf("Filas: ");
			scanf("%i", &filas);
			printf("Columnas: ");
			scanf("%i", &columnas);

			comprobar_dimensiones(filas, columnas, dimensiones_adecuadas);

			if (!dimensiones_adecuadas)
			{
				printf("Error, el tablero es demasiado grande para correr en un bloque SM\n");
				system("pause");
				system("cls");
			}
		} while (!dimensiones_adecuadas);

		juego(filas, columnas, dificultad, false, datos_partida, archivo_matriz);
	}
	else if (opcion == 2)
	{
		cargar_datos(dificultad, filas, columnas, datos_partida);
		juego(filas, columnas, dificultad, true, datos_partida, archivo_matriz);
	}

	system("pause");
	return 0;
}

void juego(int filas, int columnas, int dificultad, bool cargar_partida, FILE *& datos_partida, FILE *& archivo_matriz)
{

	bool salir = false;
	int opcion = 0;
	int num_colores = (dificultad == 1) ? 5 : 6;
	long tam_matriz = filas * columnas;
	int  * matriz = (int *)malloc(tam_matriz * sizeof(int));
	int pos_fila = 0;
	int pos_columna = 0;
	int tipo_bomba = 0;

	if (cargar_partida)
	{
		cargar_matriz(matriz, tam_matriz, archivo_matriz);
	}
	else
	{
		generar_matriz(matriz, tam_matriz, num_colores);
	}

	while (!salir)
	{
		//system("cls");
		generarAleatorios(matriz, tam_matriz, num_colores);
		printf("dificuldad = %i\tfilas = %i\tcolumnas = %i\n", dificultad, filas, columnas);
		printf(" ----------------------\n"
			"| 1 = seguir jugando   |\n"
			"| 2 = guardar partida  |\n"
			"| 3 = salir del juego  |\n"
			" ---------------------- \n\n");
		dibujar_matriz(matriz, filas, columnas);

		printf("Elija opcion: ");
		scanf("%i", &opcion);

		switch (opcion)
		{
		case 1:
		{
				  printf("Introduzca posicion de la casilla (fila/columna):\n");
				  printf("fila: ");
				  scanf("%i", &pos_fila);
				  printf("columna: ");
				  scanf("%i", &pos_columna);

				  if (es_bomba(matriz, pos_fila, pos_columna, columnas, tipo_bomba))
				  {
					  switch (tipo_bomba)
					  {
					  case BOMBAVER:
					  {
									   explotar_vertical(matriz, tam_matriz, filas, columnas, pos_columna);
									   break;
					  }
					  case BOMBAHOR:
					  {
									   explotar_horizontal(matriz, tam_matriz, filas, columnas, pos_fila);
									   break;
					  }
					  case BOMBATNT:
					  {
									   explotar_tnt(matriz, tam_matriz, filas, columnas, pos_fila, pos_columna);
									   break;
					  }
					  case BOMBAPUZZLE:
					  {
										  break;
					  }
					  default:
						  break;
					  }
				  }
				  else
				  {
					  jugar(matriz, filas, columnas, filas*columnas*sizeof(int), pos_fila, pos_columna, num_colores);
				  }

				  break;
		}
		case 2:
		{
				  guardar_partida(matriz, dificultad, filas, columnas, archivo_matriz, datos_partida);
				  break;
		}
		case 3:
		{
				  salir = true;
				  break;
		}
		default:
			break;
		}
	}
}

void comprobar_dimensiones(int filas, int columnas, bool & dimensiones_adecuadas)
{
	hipDeviceProp_t propiedades_gpu;
	hipGetDeviceProperties(&propiedades_gpu, 0);

	long capacidad_bloque = propiedades_gpu.maxThreadsPerBlock * 10000;
	long tam_matriz = filas * columnas;

	dimensiones_adecuadas = (tam_matriz > capacidad_bloque) ? false : true;
}

void generar_matriz(int *& matriz, long tam_matriz, int num_colores)
{
	for (int i = 0; i < tam_matriz; i++)
	{
		matriz[i] = rand() % num_colores + 1;
	}
}

void generarAleatorios(int *& matriz, int tam_matriz, int num_colores){
	for (int i = 0; i < tam_matriz; i++)
	{
		if (matriz[i] == 0){
			matriz[i] = rand() % num_colores + 1;
		}
	}
}

void dibujar_matriz(int * matriz, int filas, int columnas)
{
	int valor = 0;

	printf(" \t");

	for (int i = 0; i < columnas; i++)
	{
		printf("%i   ", i);
	}
	printf("\n\n\n");

	for (int i = 0; i < filas; i++)
	{
		printf("%i\t", i);
		for (int n = 0; n < columnas; n++)
		{
			valor = matriz[i * columnas + n];

			switch (valor)
			{
			case 0:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 0);
				break;
			case 1:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 1);
				break;
			case 2:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 4);
				break;
			case 3:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 13);
				break;
			case 4:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 2);
				break;
			case 5:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 6);
				break;
			case 6:
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 14);
				break;
			default:
			{
					   SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
					   break;
			}
			}

			printf("%i   ", valor);
		}
		printf("\n");
		SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), 15);
	}

}

void guardar_partida(int * matriz, int dificultad, int filas, int columnas, FILE *& archivo_matriz, FILE *& datos_partida)
{
	long tam_matriz = filas * columnas;

	if ((datos_partida = fopen("datos_partida.txt", "w")) == NULL)
	{
		printf("error abriendo el archivo \"datos_partida.txt\" \n");
		system("pause");
		exit(1);
	}
	else
	{
		fprintf(datos_partida, "%i\n", dificultad);
		fprintf(datos_partida, "%i\n", filas);
		fprintf(datos_partida, "%i", columnas);
		fclose(datos_partida);
	}

	if ((archivo_matriz = fopen("matriz.data", "wb")) == NULL)
	{
		printf("error abriendo el archivo \"archivo_matriz.txt\" \n");
		system("pause");
		exit(1);
	}
	else
	{
		fwrite(matriz, sizeof(int), tam_matriz, archivo_matriz);
		fclose(archivo_matriz);
	}
}



void cargar_datos(int &dificultad, int &filas, int &columnas, FILE *& datos_partida)
{
	if ((datos_partida = fopen("datos_partida.txt", "r")))
	{
		int linea_actual = 0;
		while (!feof(datos_partida))
		{
			linea_actual++;

			switch (linea_actual)
			{
			case 1:
			{
					  fscanf(datos_partida, "%d", &dificultad);
					  break;
			}
			case 2:
			{
					  fscanf(datos_partida, "%d", &filas);
					  break;
			}
			case 3:
			{
					  fscanf(datos_partida, "%d", &columnas);
					  break;
			}
			default:
				break;
			}
		}

		fclose(datos_partida);
	}
	else
	{
		printf("error abriendo el archivo datos_partida.txt\n");
		return;
	}
}

void cargar_matriz(int *& matriz, long tam_matriz, FILE *& archivo_matriz)
{
	if ((archivo_matriz = fopen("matriz.data", "rb")))
	{
		fread(matriz, sizeof(int), tam_matriz, archivo_matriz);
	}
	else
	{
		printf("error abriendo el archivo \"archivo_matriz\"");
		return;
	}
	fclose(archivo_matriz);
}

bool es_bomba(int * matriz, int fila, int columna, int num_columnas, int &tipo_bomba)
{
	bool es_bomba = false;
	int valor = matriz[fila * num_columnas + columna];

	if ((valor == BOMBAHOR) || (valor == BOMBAVER) || (valor == BOMBATNT) || (valor == BOMBAPUZZLE))
	{
		es_bomba = true;
		tipo_bomba = valor;
	}

	return es_bomba;
}

//
void jugar(int *tablero, int fil, int col, int size, int fila, int columna, int num_colores){
	//Este m�todo lanzar� el kernel de juego.
	//Primero creamos la variable que va al device:
	int* tableroD;
	//Reservamos memoria 
	hipMalloc(&tableroD, size);
	//Copiamos nuestro tablero al device.
	hipMemcpy(tableroD, tablero, size, hipMemcpyHostToDevice);
	dim3 DimGrid((fil + TILE_WIDTH - 1) / TILE_WIDTH, (col + TILE_WIDTH - 1) / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);
	KernelJugar << <DimGrid, DimBlock >> >(tableroD, fil, col, fila, columna, (rand() % 2 + 1), (rand() % num_colores + 1));
	hipMemcpy(tablero, tableroD, size, hipMemcpyDeviceToHost);
	hipFree(tableroD);
	//Falta mostrarlo
	dibujar_matriz(tablero, fil, col);
	hipDeviceReset();
}

void explotar_vertical(int *& tablero, long tam_tablero, int filas, int columnas, int columna)
{
	int * tablero_d;

	//reservar memoria
	hipMalloc(&tablero_d, tam_tablero * sizeof(int));
	//copiar tablero al device
	hipMemcpy(tablero_d, tablero, tam_tablero * sizeof(int), hipMemcpyHostToDevice);
	//definir tama�o de grid y de bloque
	dim3 DimGrid((filas + TILE_WIDTH - 1) / TILE_WIDTH, (columnas + TILE_WIDTH - 1) / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

	explosion_vertical << <DimGrid, DimBlock >> >(tablero_d, columnas, columna);

	//copiar tablero al host
	hipMemcpy(tablero, tablero_d, tam_tablero * sizeof(int), hipMemcpyDeviceToHost);

	dibujar_matriz(tablero, filas, columnas);

	//liberar memoria
	hipFree(tablero_d);
	hipDeviceReset();

}

void explotar_horizontal(int *& tablero, long tam_tablero, int filas, int columnas, int fila)
{
	int * tablero_d;

	//reservar memoria
	hipMalloc(&tablero_d, tam_tablero * sizeof(int));
	//copiar tablero al device
	hipMemcpy(tablero_d, tablero, tam_tablero * sizeof(int), hipMemcpyHostToDevice);
	//definir tama�o de grid y de bloque
	dim3 DimGrid((filas + TILE_WIDTH - 1) / TILE_WIDTH, (columnas + TILE_WIDTH - 1) / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

	explosion_horizontal << <DimGrid, DimBlock >> >(tablero_d, columnas, fila);

	//copiar tablero al host
	hipMemcpy(tablero, tablero_d, tam_tablero * sizeof(int), hipMemcpyDeviceToHost);

	dibujar_matriz(tablero, filas, columnas);

	//liberar memoria
	hipFree(tablero_d);
	hipDeviceReset();

}

void explotar_tnt(int *& tablero, long tam_tablero, int filas, int columnas, int fila, int columna)
{
	int * tablero_d;

	//reservar memoria
	hipMalloc(&tablero_d, tam_tablero * sizeof(int));
	//copiar tablero al device
	hipMemcpy(tablero_d, tablero, tam_tablero * sizeof(int), hipMemcpyHostToDevice);
	//definir tama�o de grid y de bloque
	dim3 DimGrid((filas + TILE_WIDTH - 1) / TILE_WIDTH, (columnas + TILE_WIDTH - 1) / TILE_WIDTH);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH);

	explosion_tnt << <DimGrid, DimBlock >> >(tablero_d, tam_tablero, filas, columnas, fila, columna);

	//copiar tablero al host
	hipMemcpy(tablero, tablero_d, tam_tablero * sizeof(int), hipMemcpyDeviceToHost);

	dibujar_matriz(tablero, filas, columnas);

	//liberar memoria
	hipFree(tablero_d);
	hipDeviceReset();

}

//KernelJugar << <DimGrid, DimBlock >> >(tableroD, fil, col, fila, columna, (rand() % 2 + 1), (rand() % num_colores + 1));
__global__ void KernelJugar(int *tablero, int fila, int columna, int i, int j, int bomba, int color){ //fila y columna indican el m�ximo n�mero en el tablero de juego, i y j las cordenadas del a eliminar.
	//Si el hilo es el que ha seleccionado el jugador:
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int numCeros = 0;
	//Comprobamos primero que no sea una bomba.
	if (x == i && y == j){
		/*if (tablero[x*columna + y] == 7 ){
		bombaHorizontal(tablero, x, y, fila, columna);
		}
		else if (tablero[x*columna + y] == 8){
		bombaVertical(tablero, x, y, fila, columna);
		}
		else if (tablero[x*columna + y] == 9){
		bombaTNT(tablero, x, y, fila, columna);
		}
		else*/ if ((tablero[x*columna + y]) == 10) {
			//printf("Entro aqu� \n");
			bombaPuzzle(tablero, x, y, fila, columna, color);
		}
		else{
			//Ejecutar� el comprobar los bloques
			printf("Estoy aqu�\n");
			comprobarBloques(tablero, x, y, fila, columna);
			for (int l = 0; l < fila*columna; l++)
			{
				if (tablero[l] == 0){
					numCeros++;
				}
			}
			printf("Num ceros: %d \n", numCeros);
			if (numCeros >= 7){
				tablero[x*fila + y] = 10;
			}
			else if (numCeros == 6){
				tablero[x*fila + y] = 9;
			}
			else if (numCeros >= 4){
				if (bomba == 1){
					tablero[x*columna + y] = 7;
				}
				else{
					tablero[x*columna + y] = 8;
				}
			}
		}
	}
	__syncthreads();

	/*for (int i = 0; i <= fila; i++){
		if (x > 0){
		if (tablero[x*columna + y] == 0 && !tablero[(x - 1)*columna + y] == 0){
		tablero[x*columna + y] = tablero[(x - 1)*columna + y];
		tablero[(x - 1)*columna + y] = 0;
		}
		}
		__syncthreads();
		}*/
	if (y < columna && x < fila) {
		if (y < columna && x < fila) {
			for (int i = 1; i < fila; i++) {

				if (tablero[(fila - i)*columna + y] == 0) {
					if (tablero[(fila - (i + 1))*columna + y] == 0) {
						int j = i;
						while (tablero[(fila - (j + 1))*columna + y] == 0 && j < fila) {
							j++;
						}
						tablero[(fila - i)*columna + y] = tablero[(fila - (j + 1))*columna + y];
						tablero[(fila - (j + 1))*columna + y] = 0;
					}
					else {
						tablero[(fila - i)*columna + y] = tablero[(fila - (i + 1))*columna + y];
						tablero[(fila - (i + 1))*columna + y] = 0;
					}
				}
				__syncthreads();

			}
		}

	}
}


__device__ void comprobarBloques(int *tablero, int x, int y, int fila, int columna){ //X indica la fila, Y la columna
	//Primero compruebo si en alg�n lateral del tablero En el juego solo puede estar arriba, abajo, derecha o izquierda.Sin diagonales.
	bool fallo = true;
	if (x != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - columna]){//Compruebo arriba
		//La primera comprobaci�n comprueba que no sea la ficha de m�s arriba, por que si lo es no puede comprobar.
		//Si se cumple es que hay una ficha igual arriba.
		fallo = false;
		comprobarBloquesArriba(tablero, x - 1, y, fila, columna);
	}
	if (y != columna - 1 && (y + 1) && tablero[(x*columna) + y] == tablero[((x*columna) + y) + 1]){//Compruebo a la derecha.
		//La primera comprobacion mira si el elemento no es el �ltimo de la matriz a la derecha, porque si lo fuera no puede comprobar a la derecha, pues 
		//Ya no habr�a m�s derecha.
		//Si se cumple.
		fallo = false;
		//Llamo a eliminar derecha.
		comprobarBloquesDerecha(tablero, x, y + 1, fila, columna);
	}
	if (y != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - 1]){ //Compruebo a la izquierda.
		//Si la columna es 0 es que es el elemento de m�s a la izquierda.
		//Si se cumple llamo a eliminar izquierda.
		fallo = false;
		comprobarBloquesIzquierda(tablero, x, y - 1, fila, columna);
	}
	if (x != fila - 1 && tablero[(x*columna) + y] == tablero[((x*columna) + y) + columna]){//Compruebo abajo.
		//La primera comprobaci�n comprubea si no es el elemento de la �ltima fila, en caso afirmativo, no busca m�s abajo pues no hay.
		//Si se cumple llamo a eliminar abajo.
		fallo = false;
		comprobarBloquesAbajo(tablero, x + 1, y, fila, columna);
	}
	if (fallo == true){
		printf("Ninguna combinaci�n posible,vuelve a intentarlo:");
	}
	else{
		tablero[(x*columna) + y] = 0;//Lo pasamos a cero para despu�s eliminarlo.
	}
}

__device__ void comprobarBloquesArriba(int *tablero, int x, int y, int fila, int columna){
	//Funci�n que comprueba arriba del bloque inicial si hay m�s bloques a eliminar.
	//Misma comprobaci�n que en comprobar bloques normales solo que ya no mira abajo.
	if (x != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - columna]){//Compruebo arriba
		//Si se cumple es que hay una ficha igual arriba.
		comprobarBloquesArriba(tablero, x - 1, y, fila, columna);
	}
	if (y != columna - 1 && (y + 1) && tablero[(x*columna) + y] == tablero[((x*columna) + y) + 1]){//Compruebo a la derecha.
		//La primera comprobacion mira si el elemento no es el �ltimo de la matriz a la derecha, porque si lo fuera no puede comprobar a la derecha, pues 
		//Ya no habr�a m�s derecha.
		//Si se cumple.
		//Llamo a eliminar derecha.
		comprobarBloquesDerecha(tablero, x, y + 1, fila, columna);
	}
	if (y != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - 1]){ //Compruebo a la izquierda.
		//Si la columna es 0 es que es el elemento de m�s a la izquierda.
		//Si se cumple llamo a eliminar izquierda.
		comprobarBloquesIzquierda(tablero, x, y - 1, fila, columna);
	}
	tablero[(x*columna) + y] = 0;
}

__device__ void comprobarBloquesDerecha(int *tablero, int x, int y, int fila, int columna){
	//Funci�n que comprueba a la derecha del bloque inicial si hay m�s bloques a eliminar.

	//Misma comprobaci�n que en comprobar bloques normales solo que ya no mira a la izquierda.
	if (x != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - columna]){//Compruebo arriba
		//Si se cumple es que hay una ficha igual arriba.
		comprobarBloquesArriba(tablero, x - 1, y, fila, columna);
	}
	if (y != columna - 1 && (y + 1) && tablero[(x*columna) + y] == tablero[((x*columna) + y) + 1]){//Compruebo a la derecha.
		//La primera comprobacion mira si el elemento no es el �ltimo de la matriz a la derecha, porque si lo fuera no puede comprobar a la derecha, pues 
		//Ya no habr�a m�s derecha.
		//Si se cumple.
		comprobarBloquesDerecha(tablero, x, y + 1, fila, columna);
		//Llamo a eliminar derecha.
	}
	if (x != fila - 1 && tablero[(x*columna) + y] == tablero[((x*columna) + y) + columna]){//Compruebo abajo.
		//La primera comprobaci�n comprubea si no es el elemento de la �ltima fila, en caso afirmativo, no busca m�s abajo pues no hay.
		//Si se cumple llamo a eliminar abajo.
		comprobarBloquesAbajo(tablero, x + 1, y, fila, columna);
	}
	tablero[(x*columna) + y] = 0;//Si se llama a esta funci�n, es que el elemento actual tambi�n debemos eliminarlo.
}

__device__ void comprobarBloquesIzquierda(int *tablero, int x, int y, int fila, int columna){
	//Funci�n que comprueba a la izquierda del bloque inicial si hay m�s bloques a eliminar.
	//Misma comprobaci�n que en comprobar bloques normales solo que ya no mira a la derecha.
	if (x != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - columna]){//Compruebo arriba
		//Si se cumple es que hay una ficha igual arriba.
		comprobarBloquesArriba(tablero, x - 1, y, fila, columna);

	}
	if (y != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - 1]){ //Compruebo a la izquierda.
		//Si la columna es 0 es que es el elemento de m�s a la izquierda.
		//Si se cumple llamo a eliminar izquierda.
		comprobarBloquesIzquierda(tablero, x, y - 1, fila, columna);
	}
	if (x != fila - 1 && tablero[(x*columna) + y] == tablero[((x*columna) + y) + columna]){//Compruebo abajo.
		//La primera comprobaci�n comprubea si no es el elemento de la �ltima fila, en caso afirmativo, no busca m�s abajo pues no hay.
		//Si se cumple llamo a eliminar abajo.
		comprobarBloquesAbajo(tablero, x + 1, y, fila, columna);
	}
	tablero[(x*columna) + y] = 0;//Si se llama a esta funci�n, es que el elemento actual tambi�n debemos eliminarlo.
}

__device__ void comprobarBloquesAbajo(int *tablero, int x, int y, int fila, int columna){
	//Funci�n que comprueba abajo del bloque inicial si hay m�s bloques a eliminar.
	//Misma comprobaci�n que en comprobar bloques normales solo que ya no mira arriba
	if (y != columna - 1 && (y + 1) && tablero[(x*columna) + y] == tablero[((x*columna) + y) + 1]){//Compruebo a la derecha.
		//La primera comprobacion mira si el elemento no es el �ltimo de la matriz a la derecha, porque si lo fuera no puede comprobar a la derecha, pues 
		//Ya no habr�a m�s derecha.
		//Si se cumple.
		comprobarBloquesDerecha(tablero, x, y + 1, fila, columna);
		//Llamo a eliminar derecha.
	}
	if (y != 0 && tablero[(x*columna) + y] == tablero[((x*columna) + y) - 1]){ //Compruebo a la izquierda.
		//Si la columna es 0 es que es el elemento de m�s a la izquierda.
		//Si se cumple llamo a eliminar izquierda.
		comprobarBloquesIzquierda(tablero, x, y - 1, fila, columna);
	}
	if (x != fila - 1 && tablero[(x*columna) + y] == tablero[((x*columna) + y) + columna]){//Compruebo abajo.
		//La primera comprobaci�n comprubea si no es el elemento de la �ltima fila, en caso afirmativo, no busca m�s abajo pues no hay.
		comprobarBloquesAbajo(tablero, x + 1, y, fila, columna);
		//Si se cumple llamo a eliminar abajo.
	}
	tablero[(x*columna) + y] = 0;//Si se llama a esta funci�n, es que el elemento actual tambi�n debemos eliminarlo.
}

/*__device__ void bombaVertical(int *tablero, int x, int y, int fila, int columna){
tablero[(x*columna) + y] = 0;
if (x != fila - 1){
borrarAbajo(tablero, x + 1, y, fila, columna);
}
if (x != 0){
borrarArriba(tablero, x - 1, y, fila, columna);
}
}

__device__ void bombaHorizontal(int *tablero, int x, int y, int fila, int columna){
tablero[(x*columna) + y] = 0;
if (y != columna - 1){
borrarDerecha(tablero, x, y + 1, fila, columna);
}
if (y != 0){
borrarIzquierda(tablero, x, y - 1, fila, columna);
}
}

__device__ void bombaTNT(int *tablero, int x, int y, int fila, int columna){
tablero[(x*columna) + y] = 0;
if (x != fila - 1){//Abajo
tablero[((x + 1)*columna) + y] = 0;
//AbajoDerecha
if (y != columna - 1){
tablero[((x + 1)*columna) + (y + 1)] = 0;
}//AbajoIzquierda
if (y != 0){
tablero[((x + 1)*columna) + (y - 1)] = 0;
}

}
if (x != 0){
tablero[((x - 1)*columna) + y] = 0;
//ArribaDerecha
if (y != columna - 1){
tablero[((x - 1)*columna) + (y + 1)] = 0;
}//ArribaIzquierda
if (y != 0){
tablero[((x - 1)*columna) + (y - 1)] = 0;
}
}
if (y != columna - 1){
tablero[(x*columna) + (y + 1)] = 0;
}
if (y != 0){
tablero[(x*columna) + (y - 1)] = 0;
}

}*/
__device__ void bombaPuzzle(int *tablero, int x, int y, int fila, int columna, int color){
	tablero[(x*columna) + y] = 0;
	for (int l = 0; l < fila*columna; l++)
	{
		if (tablero[l] == color){
			printf("He puesto un color a O %d \n", color);
			tablero[l] = 0;
		}
	}
}
__device__ void borrarAbajo(int *tablero, int x, int y, int fila, int columna){
	tablero[(x*columna) + y] = 0;
	if (x + 1 <= fila - 1){//Abajo
		borrarAbajo(tablero, x + 1, y, fila, columna);
	}
}

__device__ void borrarArriba(int *tablero, int x, int y, int fila, int columna){
	tablero[(x*columna) + y] = 0;
	if (x - 1 >= 0){//Arriba
		borrarAbajo(tablero, x - 1, y, fila, columna);
	}
}

__device__ void borrarDerecha(int *tablero, int x, int y, int fila, int columna){
	tablero[(x*columna) + y] = 0;
	if (y + 1 != columna - 1){
		borrarDerecha(tablero, x, y + 1, fila, columna);
	}
}

__device__ void borrarIzquierda(int *tablero, int x, int y, int fila, int columna){
	tablero[(x*columna) + y] = 0;
	if (y - 1 != 0){
		borrarIzquierda(tablero, x, y - 1, fila, columna);
	}
}

__global__ void explosion_vertical(int * tablero, int anchura_tablero, int columna)
{
	int columna_hilo = blockIdx.x*blockDim.x + threadIdx.x;
	int fila_hilo = blockIdx.y*blockDim.y + threadIdx.y;

	if (columna_hilo == columna)
	{
		tablero[fila_hilo * anchura_tablero + columna_hilo] = 0;
	}

	__syncthreads();
}

__global__ void explosion_horizontal(int * tablero, int anchura_tablero, int fila)
{
	int columna_hilo = blockIdx.x*blockDim.x + threadIdx.x;
	int fila_hilo = blockIdx.y*blockDim.y + threadIdx.y;

	if (fila_hilo == fila)
	{
		tablero[fila_hilo * anchura_tablero + columna_hilo] = 0;
	}

	__syncthreads();
}

__global__ void explosion_tnt(int * tablero, long tam_tablero, int filas, int columnas, int fila, int columna)
{
	int columna_hilo = blockIdx.x*blockDim.x + threadIdx.x;
	int fila_hilo = blockIdx.y*blockDim.y + threadIdx.y;
	int pos_hilo = fila_hilo * columnas + columna_hilo;//posici�n del hilo

	//posicion elegida por el usuario
	int pos_elegida = fila * columnas + columna;

	//posiciones contiguas
	int arriba = (fila - 1) * columnas + columna;
	int abajo = (fila + 1) * columnas + columna;
	int derecha = fila * columnas + columna + 1;
	int izquierda = fila * columnas + columna - 1;
	int arriba_izq = (fila - 1) * columnas + columna - 1;
	int arriba_der = (fila - 1) * columnas + columna + 1;
	int abajo_izq = (fila + 1) * columnas + columna - 1;
	int abajo_der = (fila + 1) * columnas + columna + 1;

	if ((pos_hilo == pos_elegida) || (pos_hilo == arriba) || (pos_hilo == abajo) || (pos_hilo == derecha) || (pos_hilo == izquierda))
	{
		tablero[pos_hilo] = 0;
	}
	else if ((pos_hilo == arriba_izq) || (pos_hilo == abajo_izq) || (pos_hilo == arriba_der) || (pos_hilo == abajo_der))
	{
		tablero[pos_hilo] = 0;
	}
}

